#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t>
__global__ void bincount_cuda_kernel(scalar_t *__restrict__ src, int64_t *out,
                                     size_t numel) {
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (ptrdiff_t i = index; i < numel; i += stride) {
    atomicAdd(out + (ptrdiff_t)src[i], 1);
  }
}

at::Tensor bincount_cuda(at::Tensor src, int64_t size) {
  auto out = at::zeros(src.type().toScalarType(at::kLong), {size});

  AT_DISPATCH_ALL_TYPES(src.type(), "bincount_cuda_kernel", [&] {
    bincount_cuda_kernel<scalar_t><<<BLOCKS(src.numel()), THREADS>>>(
        src.data<scalar_t>(), out.data<int64_t>(), src.numel());
  });

  return out;
}
